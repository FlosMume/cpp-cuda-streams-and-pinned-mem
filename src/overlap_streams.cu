#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call) do { \
  hipError_t err__ = (call); \
  if (err__ != hipSuccess) { \
    fprintf(stderr, "CUDA error %s at %s:%d\n", hipGetErrorString(err__), __FILE__, __LINE__); \
    std::exit(EXIT_FAILURE); \
  } \
} while (0)

// Simple compute: y = a*x + b with extra flops per element to amplify kernel time
__global__ void saxpy_heavy(const float* __restrict__ x,
                            const float* __restrict__ y,
                            float* __restrict__ z,
                            int n,
                            float a, float b, int flop_iters)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    float vx = x[i], vy = y[i];
    // Burn flops
    #pragma unroll 4
    for (int k = 0; k <  flop_iters; ++k) {
      vx = a * vx + b;
      vy = a * vy + b;
    }
    z[i] = vx + vy;
  }
}

// Utility: get env var or default
static inline int getenv_int(const char* name, int defv) {
  if (const char* s = std::getenv(name)) {
    try { return std::max(1, std::stoi(s)); } catch (...) { return defv; }
  }
  return defv;
}

int main() {
  // Tunables
  const int N = getenv_int("N", 1<<24);             // total elements
  const int N_STREAMS = getenv_int("N_STREAMS", 4); // number of streams
  const int FLOP_ITERS = getenv_int("FLOP_ITERS", 256);

  printf("Config: N=%d (%.2f MiB/vec)  N_STREAMS=%d  FLOP_ITERS=%d\n",
         N, (N * sizeof(float)) / (1024.0f*1024.0f), N_STREAMS, FLOP_ITERS);

  // Partition into equal-size chunks; last chunk may carry remainder
  int chunk_elems = (N + N_STREAMS - 1) / N_STREAMS;

  // Pinned host allocations enable async DMA
  float *h_x = nullptr, *h_y = nullptr, *h_z = nullptr;
  CHECK_CUDA(hipHostMalloc(&h_x, N * sizeof(float)));
  CHECK_CUDA(hipHostMalloc(&h_y, N * sizeof(float)));
  CHECK_CUDA(hipHostMalloc(&h_z, N * sizeof(float)));

  // Initialize host data
  for (int i = 0; i < N; ++i) {
    h_x[i] = 1.0f;
    h_y[i] = 2.0f;
  }

  // Device buffers sized to one chunk per stream (double-buffering not required here)
  float *d_x = nullptr, *d_y = nullptr, *d_z = nullptr;
  CHECK_CUDA(hipMalloc(&d_x, chunk_elems * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_y, chunk_elems * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_z, chunk_elems * sizeof(float)));

  // Create streams and events
  std::vector<hipStream_t> streams(N_STREAMS); // Create a dynamic array (vector) that stores elements of type hipStream_t
  for (int s = 0; s < N_STREAMS; ++s) CHECK_CUDA(hipStreamCreate(&streams[s]));

  hipEvent_t start_all, stop_all;
  CHECK_CUDA(hipEventCreate(&start_all));
  CHECK_CUDA(hipEventCreate(&stop_all));

  // Baseline (single stream, synchronous pipeline)
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipEventRecord(start_all));

  const float a = 1.0001f, b = 0.0001f;
  const int block = 256;

  for (int off = 0; off < N; off += chunk_elems) {
    int this_elems = min(chunk_elems, N - off);
    int grid = (this_elems + block - 1) / block;

    CHECK_CUDA(hipMemcpy(d_x, h_x + off, this_elems * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y, h_y + off, this_elems * sizeof(float), hipMemcpyHostToDevice));

    saxpy_heavy<<<grid, block>>>(d_x, d_y, d_z, this_elems, a, b, FLOP_ITERS);
    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipMemcpy(h_z + off, d_z, this_elems * sizeof(float), hipMemcpyDeviceToHost));
  }

  CHECK_CUDA(hipEventRecord(stop_all));
  CHECK_CUDA(hipEventSynchronize(stop_all));
  float ms_baseline = 0.f;
  CHECK_CUDA(hipEventElapsedTime(&ms_baseline, start_all, stop_all));

  // Validate
  double checksum_base = 0.0;
  for (int i = 0; i < N; ++i) checksum_base += h_z[i];

  // Overlapped pipeline using multiple streams
  CHECK_CUDA(hipMemset(h_z, 0, N*sizeof(float))); // reuse output

  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipEventRecord(start_all));

  for (int off = 0, s = 0; off < N; off += chunk_elems, s = (s + 1) % N_STREAMS) {
    int this_elems = min(chunk_elems, N - off);
    int grid = (this_elems + block - 1) / block;
    hipStream_t st = streams[s];

    CHECK_CUDA(hipMemcpyAsync(d_x + s*0, h_x + off, this_elems * sizeof(float), hipMemcpyHostToDevice, st));
    CHECK_CUDA(hipMemcpyAsync(d_y + s*0, h_y + off, this_elems * sizeof(float), hipMemcpyHostToDevice, st));

    saxpy_heavy<<<grid, block, 0, st>>>(d_x + 0, d_y + 0, d_z + 0, this_elems, a, b, FLOP_ITERS);
    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipMemcpyAsync(h_z + off, d_z + 0, this_elems * sizeof(float), hipMemcpyDeviceToHost, st));
  }

  // Sync all streams
  for (int s = 0; s < N_STREAMS; ++s) CHECK_CUDA(hipStreamSynchronize(streams[s]));

  CHECK_CUDA(hipEventRecord(stop_all));
  CHECK_CUDA(hipEventSynchronize(stop_all));
  float ms_overlap = 0.f;
  CHECK_CUDA(hipEventElapsedTime(&ms_overlap, start_all, stop_all));

  // Validate (coarse)
  double checksum_ovl = 0.0;
  for (int i = 0; i < N; ++i) checksum_ovl += h_z[i];

  // Report
  double bytes = 3.0 * N * sizeof(float); // H2D x2 + D2H
  double GB = bytes / 1e9;
  double bw_base = GB / (ms_baseline / 1e3);
  double bw_ovl  = GB / (ms_overlap / 1e3);

  printf("\nBaseline (1 stream):  %.3f ms,  %.2f GB moved → %.2f GB/s  checksum=%.6e\n",
         ms_baseline, GB, bw_base, checksum_base);
  printf("Overlap  (%d streams): %.3f ms,  %.2f GB moved → %.2f GB/s  checksum=%.6e\n",
         N_STREAMS, ms_overlap, GB, bw_ovl, checksum_ovl);
  printf("Speedup: %.2fx\n", ms_baseline / ms_overlap);

  // Cleanup
  for (int s = 0; s < N_STREAMS; ++s) hipStreamDestroy(streams[s]);
  hipEventDestroy(start_all);
  hipEventDestroy(stop_all);
  hipFree(d_x); hipFree(d_y); hipFree(d_z);
  hipHostFree(h_x); hipHostFree(h_y); hipHostFree(h_z);
  return 0;
}
